#include "hip/hip_runtime.h"
/*
icc propagate-toz-test.C -o propagate-toz-test.exe -fopenmp -O3
*/
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>

#include <chrono>
#include <iomanip>
#include <iostream>

#ifndef nevts
#define nevts 100
#endif
#ifndef bsize
#define bsize 128
#endif
#ifndef ntrks
#define ntrks 9600 //122880
#endif

#define nb    ntrks/bsize
#define smear 0.1

#ifndef NITER
#define NITER 5 
#endif
#ifndef nlayer
#define nlayer 20
#endif
#ifndef num_streams
#define num_streams 7 //streams changes answers
#endif

#ifndef threadsperblockx
#define threadsperblockx 32
#endif
#define threadsperblocky 512/threadsperblockx
//#define threadsperblocky 1024/threadsperblockx  //unclear why bit 1024 total threads per block gives resource error when running with more than one layer
#ifndef blockspergrid
#define blockspergrid 15
#endif


#define HOSTDEV __host__ __device__

HOSTDEV size_t PosInMtrx(size_t i, size_t j, size_t D) {
  return i*D+j;
}

HOSTDEV size_t SymOffsets33(size_t i) {
  const size_t offs[9] = {0, 1, 3, 1, 2, 4, 3, 4, 5};
  return offs[i];
}

HOSTDEV size_t SymOffsets66(size_t i) {
  const size_t offs[36] = {0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};
  return offs[i];
}


struct ATRK {
  float par[6];
  float cov[21];
  int q;
  int hitidx[22];
};

struct AHIT {
  float pos[3];
  float cov[6];
};

struct MP1I {
  int data[1*bsize];
};

struct MP22I {
  int data[22*bsize];
};

struct MP3F {
  float data[3*bsize];
};

struct MP6F {
  float data[6*bsize];
};

struct MP3x3 {
  float data[9*bsize];
};
struct MP3x6 {
  float data[18*bsize];
};

struct MP3x3SF {
  float data[6*bsize];
};

struct MP6x6SF {
  float data[21*bsize];
};

struct MP6x6F {
  float data[36*bsize];
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;
  MP22I   hitidx;
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
};



float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0);
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
  call = !call;
  return (mu + sigma * (float) X1);
}

MPTRK* prepareTracks(ATRK inputtrk) {
  MPTRK* result;
  hipMallocManaged((void**)&result,nevts*nb*sizeof(MPTRK)); //fixme, align?
  hipMemAdvise(result,nevts*nb*sizeof(MPTRK),hipMemAdviseSetPreferredLocation,hipCpuDeviceId);
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
        //par
        for (size_t ip=0;ip<6;++ip) {
          result[ib + nb*ie].par.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.par[ip];
        }
        //cov
        for (size_t ip=0;ip<21;++ip) {
          result[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.cov[ip];
        }
        //q
        result[ib + nb*ie].q.data[it] = inputtrk.q-2*ceil(-0.5 + (float)rand() / RAND_MAX);//fixme check
      }
    }
  }
  return result;
}

MPHIT* prepareHits(AHIT inputhit) {
  //MPHIT* result = (MPHIT*) malloc(nevts*nb*sizeof(MPHIT));
  MPHIT* result;
  hipMallocManaged((void**)&result,nlayer*nevts*nb*sizeof(MPHIT));  //fixme, align?
  hipMemAdvise(result,nlayer*nevts*nb*sizeof(MPHIT),hipMemAdviseSetPreferredLocation,hipCpuDeviceId);
  for (int lay=0;lay<nlayer;++lay) {
    for (size_t ie=0;ie<nevts;++ie) {
      for (size_t ib=0;ib<nb;++ib) {
        for (size_t it=0;it<bsize;++it) {
          //pos
          for (size_t ip=0;ip<3;++ip) {
            result[lay+nlayer*(ib + nb*ie)].pos.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.pos[ip];
          }
          //cov
          for (size_t ip=0;ip<6;++ip) {
            result[lay+nlayer*(ib + nb*ie)].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.cov[ip];
          }
        }
      }
    }
  }
  return result;
}


HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib,int layer) {
  return &(tracks[ib + nb*ev+layer*nevts]);
}
HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib,int layer) {
  return &(tracks[ib + nb*ev+layer*nevts]);
}
HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}


HOSTDEV float q(const MP1I* bq, size_t it){
  return (*bq).data[it];
}

HOSTDEV float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it + ipar*bsize];
}
HOSTDEV float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float x    (MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
HOSTDEV float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
HOSTDEV float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
HOSTDEV float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
HOSTDEV float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
HOSTDEV float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
HOSTDEV float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }

HOSTDEV float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}

HOSTDEV float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
HOSTDEV float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
HOSTDEV float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
HOSTDEV float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
HOSTDEV float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
HOSTDEV float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }

HOSTDEV void setpar(MP6F* bpars, size_t it, size_t ipar, float val){
  (*bpars).data[it + ipar*bsize] = val;
}
HOSTDEV void setx    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 0, val); }
HOSTDEV void sety    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 1, val); }
HOSTDEV void setz    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 2, val); }
HOSTDEV void setipt  (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 3, val); }
HOSTDEV void setphi  (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 4, val); }
HOSTDEV void settheta(MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 5, val); }

HOSTDEV void setpar(MPTRK* btracks, size_t it, size_t ipar, float val){
  return setpar(&(*btracks).par,it,ipar,val);
}
HOSTDEV void setx    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 0, val); }
HOSTDEV void sety    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 1, val); }
HOSTDEV void setz    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 2, val); }
HOSTDEV void setipt  (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 3, val); }
HOSTDEV void setphi  (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 4, val); }
HOSTDEV void settheta(MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 5, val); }

HOSTDEV MPHIT* bHit(MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib,int lay) {
  return &(hits[lay + (ib*nlayer) +(ev*nlayer*nb)]);
}

HOSTDEV float pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it + ipar*bsize];
}
HOSTDEV float x(const MP3F* hpos, size_t it)    { return pos(hpos, it, 0); }
HOSTDEV float y(const MP3F* hpos, size_t it)    { return pos(hpos, it, 1); }
HOSTDEV float z(const MP3F* hpos, size_t it)    { return pos(hpos, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t it, size_t ipar){
  return pos(&(*hits).pos,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t it)    { return pos(hits, it, 0); }
HOSTDEV float y(const MPHIT* hits, size_t it)    { return pos(hits, it, 1); }
HOSTDEV float z(const MPHIT* hits, size_t it)    { return pos(hits, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPHIT* bhits = bHit(hits, ev, ib);
  size_t it = tk % bsize;
  return pos(bhits,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 0); }
HOSTDEV float y(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 1); }
HOSTDEV float z(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 2); }



#define N bsize
__forceinline__ __device__ void MultHelixPropEndcap(const MP6x6F* A, const MP6x6SF* B, MP6x6F* C) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = b[ 0*N+n] + a[ 2*N+n]*b[ 3*N+n] + a[ 3*N+n]*b[ 6*N+n] + a[ 4*N+n]*b[10*N+n] + a[ 5*N+n]*b[15*N+n];
    c[ 1*N+n] = b[ 1*N+n] + a[ 2*N+n]*b[ 4*N+n] + a[ 3*N+n]*b[ 7*N+n] + a[ 4*N+n]*b[11*N+n] + a[ 5*N+n]*b[16*N+n];
    c[ 2*N+n] = b[ 3*N+n] + a[ 2*N+n]*b[ 5*N+n] + a[ 3*N+n]*b[ 8*N+n] + a[ 4*N+n]*b[12*N+n] + a[ 5*N+n]*b[17*N+n];
    c[ 3*N+n] = b[ 6*N+n] + a[ 2*N+n]*b[ 8*N+n] + a[ 3*N+n]*b[ 9*N+n] + a[ 4*N+n]*b[13*N+n] + a[ 5*N+n]*b[18*N+n];
    c[ 4*N+n] = b[10*N+n] + a[ 2*N+n]*b[12*N+n] + a[ 3*N+n]*b[13*N+n] + a[ 4*N+n]*b[14*N+n] + a[ 5*N+n]*b[19*N+n];
    c[ 5*N+n] = b[15*N+n] + a[ 2*N+n]*b[17*N+n] + a[ 3*N+n]*b[18*N+n] + a[ 4*N+n]*b[19*N+n] + a[ 5*N+n]*b[20*N+n];
    c[ 6*N+n] = b[ 1*N+n] + a[ 8*N+n]*b[ 3*N+n] + a[ 9*N+n]*b[ 6*N+n] + a[10*N+n]*b[10*N+n] + a[11*N+n]*b[15*N+n];
    c[ 7*N+n] = b[ 2*N+n] + a[ 8*N+n]*b[ 4*N+n] + a[ 9*N+n]*b[ 7*N+n] + a[10*N+n]*b[11*N+n] + a[11*N+n]*b[16*N+n];
    c[ 8*N+n] = b[ 4*N+n] + a[ 8*N+n]*b[ 5*N+n] + a[ 9*N+n]*b[ 8*N+n] + a[10*N+n]*b[12*N+n] + a[11*N+n]*b[17*N+n];
    c[ 9*N+n] = b[ 7*N+n] + a[ 8*N+n]*b[ 8*N+n] + a[ 9*N+n]*b[ 9*N+n] + a[10*N+n]*b[13*N+n] + a[11*N+n]*b[18*N+n];
    c[10*N+n] = b[11*N+n] + a[ 8*N+n]*b[12*N+n] + a[ 9*N+n]*b[13*N+n] + a[10*N+n]*b[14*N+n] + a[11*N+n]*b[19*N+n];
    c[11*N+n] = b[16*N+n] + a[ 8*N+n]*b[17*N+n] + a[ 9*N+n]*b[18*N+n] + a[10*N+n]*b[19*N+n] + a[11*N+n]*b[20*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = 0;
    c[14*N+n] = 0;
    c[15*N+n] = 0;
    c[16*N+n] = 0;
    c[17*N+n] = 0;
    c[18*N+n] = b[ 6*N+n];
    c[19*N+n] = b[ 7*N+n];
    c[20*N+n] = b[ 8*N+n];
    c[21*N+n] = b[ 9*N+n];
    c[22*N+n] = b[13*N+n];
    c[23*N+n] = b[18*N+n];
    c[24*N+n] = a[26*N+n]*b[ 3*N+n] + a[27*N+n]*b[ 6*N+n] + b[10*N+n] + a[29*N+n]*b[15*N+n];
    c[25*N+n] = a[26*N+n]*b[ 4*N+n] + a[27*N+n]*b[ 7*N+n] + b[11*N+n] + a[29*N+n]*b[16*N+n];
    c[26*N+n] = a[26*N+n]*b[ 5*N+n] + a[27*N+n]*b[ 8*N+n] + b[12*N+n] + a[29*N+n]*b[17*N+n];
    c[27*N+n] = a[26*N+n]*b[ 8*N+n] + a[27*N+n]*b[ 9*N+n] + b[13*N+n] + a[29*N+n]*b[18*N+n];
    c[28*N+n] = a[26*N+n]*b[12*N+n] + a[27*N+n]*b[13*N+n] + b[14*N+n] + a[29*N+n]*b[19*N+n];
    c[29*N+n] = a[26*N+n]*b[17*N+n] + a[27*N+n]*b[18*N+n] + b[19*N+n] + a[29*N+n]*b[20*N+n];
    c[30*N+n] = b[15*N+n];
    c[31*N+n] = b[16*N+n];
    c[32*N+n] = b[17*N+n];
    c[33*N+n] = b[18*N+n];
    c[34*N+n] = b[19*N+n];
    c[35*N+n] = b[20*N+n];
  }
}

__forceinline__ __device__ void MultHelixPropTranspEndcap(MP6x6F* A, MP6x6F* B, MP6x6SF* C) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = b[ 0*N+n] + b[ 2*N+n]*a[ 2*N+n] + b[ 3*N+n]*a[ 3*N+n] + b[ 4*N+n]*a[ 4*N+n] + b[ 5*N+n]*a[ 5*N+n];
    c[ 1*N+n] = b[ 6*N+n] + b[ 8*N+n]*a[ 2*N+n] + b[ 9*N+n]*a[ 3*N+n] + b[10*N+n]*a[ 4*N+n] + b[11*N+n]*a[ 5*N+n];
    c[ 2*N+n] = b[ 7*N+n] + b[ 8*N+n]*a[ 8*N+n] + b[ 9*N+n]*a[ 9*N+n] + b[10*N+n]*a[10*N+n] + b[11*N+n]*a[11*N+n];
    c[ 3*N+n] = b[12*N+n] + b[14*N+n]*a[ 2*N+n] + b[15*N+n]*a[ 3*N+n] + b[16*N+n]*a[ 4*N+n] + b[17*N+n]*a[ 5*N+n];
    c[ 4*N+n] = b[13*N+n] + b[14*N+n]*a[ 8*N+n] + b[15*N+n]*a[ 9*N+n] + b[16*N+n]*a[10*N+n] + b[17*N+n]*a[11*N+n];
    c[ 5*N+n] = 0;
    c[ 6*N+n] = b[18*N+n] + b[20*N+n]*a[ 2*N+n] + b[21*N+n]*a[ 3*N+n] + b[22*N+n]*a[ 4*N+n] + b[23*N+n]*a[ 5*N+n];
    c[ 7*N+n] = b[19*N+n] + b[20*N+n]*a[ 8*N+n] + b[21*N+n]*a[ 9*N+n] + b[22*N+n]*a[10*N+n] + b[23*N+n]*a[11*N+n];
    c[ 8*N+n] = 0;
    c[ 9*N+n] = b[21*N+n];
    c[10*N+n] = b[24*N+n] + b[26*N+n]*a[ 2*N+n] + b[27*N+n]*a[ 3*N+n] + b[28*N+n]*a[ 4*N+n] + b[29*N+n]*a[ 5*N+n];
    c[11*N+n] = b[25*N+n] + b[26*N+n]*a[ 8*N+n] + b[27*N+n]*a[ 9*N+n] + b[28*N+n]*a[10*N+n] + b[29*N+n]*a[11*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = b[27*N+n];
    c[14*N+n] = b[26*N+n]*a[26*N+n] + b[27*N+n]*a[27*N+n] + b[28*N+n] + b[29*N+n]*a[29*N+n];
    c[15*N+n] = b[30*N+n] + b[32*N+n]*a[ 2*N+n] + b[33*N+n]*a[ 3*N+n] + b[34*N+n]*a[ 4*N+n] + b[35*N+n]*a[ 5*N+n];
    c[16*N+n] = b[31*N+n] + b[32*N+n]*a[ 8*N+n] + b[33*N+n]*a[ 9*N+n] + b[34*N+n]*a[10*N+n] + b[35*N+n]*a[11*N+n];
    c[17*N+n] = 0;
    c[18*N+n] = b[33*N+n];
    c[19*N+n] = b[32*N+n]*a[26*N+n] + b[33*N+n]*a[27*N+n] + b[34*N+n] + b[35*N+n]*a[29*N+n];
    c[20*N+n] = b[35*N+n];
  }
}

__forceinline__ __device__ void KalmanGainInv(const MP6x6SF* A, const MP3x3SF* B, MP3x3* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the inverse of HpHt +R
  const float* a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const float* b = (*B).data; //ASSUME_ALIGNED(b, 64);
  float* c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    double det =
      ((a[0*N+n]+b[0*N+n])*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])))) -
      ((a[1*N+n]+b[1*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])))) +
      ((a[2*N+n]+b[2*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n]))));
    double invdet = 1.0/det;

    c[ 0*N+n] =  invdet*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 1*N+n] =  -1*invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 2*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 3*N+n] =  -1*invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 4*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 5*N+n] =  -1*invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 6*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n])));
    c[ 7*N+n] =  -1*invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 8*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[6*N+n]+b[3*N+n])) - ((a[1*N+n]+b[1*N+n]) *(a[1*N+n]+b[1*N+n])));
  }
 // __syncthreads();
}

__forceinline__ __device__ void KalmanGain(const MP6x6SF* A, const MP3x3* B, MP3x6* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the kalman gain
  const float* a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const float* b = (*B).data; //ASSUME_ALIGNED(b, 64);
  float* c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = a[0*N+n]*b[0*N+n] + a[1*N+n]*b[3*N+n] + a[2*N+n]*b[6*N+n];
    c[ 1*N+n] = a[0*N+n]*b[1*N+n] + a[1*N+n]*b[4*N+n] + a[2*N+n]*b[7*N+n];
    c[ 2*N+n] = a[0*N+n]*b[2*N+n] + a[1*N+n]*b[5*N+n] + a[2*N+n]*b[8*N+n];
    c[ 3*N+n] = a[1*N+n]*b[0*N+n] + a[6*N+n]*b[3*N+n] + a[7*N+n]*b[6*N+n];
    c[ 4*N+n] = a[1*N+n]*b[1*N+n] + a[6*N+n]*b[4*N+n] + a[7*N+n]*b[7*N+n];
    c[ 5*N+n] = a[1*N+n]*b[2*N+n] + a[6*N+n]*b[5*N+n] + a[7*N+n]*b[8*N+n];
    c[ 6*N+n] = a[2*N+n]*b[0*N+n] + a[7*N+n]*b[3*N+n] + a[11*N+n]*b[6*N+n];
    c[ 7*N+n] = a[2*N+n]*b[1*N+n] + a[7*N+n]*b[4*N+n] + a[11*N+n]*b[7*N+n];
    c[ 8*N+n] = a[2*N+n]*b[2*N+n] + a[7*N+n]*b[5*N+n] + a[11*N+n]*b[8*N+n];
    c[ 9*N+n] = a[3*N+n]*b[0*N+n] + a[8*N+n]*b[3*N+n] + a[12*N+n]*b[6*N+n];
    c[ 10*N+n] = a[3*N+n]*b[1*N+n] + a[8*N+n]*b[4*N+n] + a[12*N+n]*b[7*N+n];
    c[ 11*N+n] = a[3*N+n]*b[2*N+n] + a[8*N+n]*b[5*N+n] + a[12*N+n]*b[8*N+n];
    c[ 12*N+n] = a[4*N+n]*b[0*N+n] + a[9*N+n]*b[3*N+n] + a[13*N+n]*b[6*N+n];
    c[ 13*N+n] = a[4*N+n]*b[1*N+n] + a[9*N+n]*b[4*N+n] + a[13*N+n]*b[7*N+n];
    c[ 14*N+n] = a[4*N+n]*b[2*N+n] + a[9*N+n]*b[5*N+n] + a[13*N+n]*b[8*N+n];
    c[ 15*N+n] = a[5*N+n]*b[0*N+n] + a[10*N+n]*b[3*N+n] + a[14*N+n]*b[6*N+n];
    c[ 16*N+n] = a[5*N+n]*b[1*N+n] + a[10*N+n]*b[4*N+n] + a[14*N+n]*b[7*N+n];
    c[ 17*N+n] = a[5*N+n]*b[2*N+n] + a[10*N+n]*b[5*N+n] + a[14*N+n]*b[8*N+n];
  }
  //__syncthreads();
}

__forceinline__ __device__ void KalmanUpdate(MP6x6SF* trkErr, MP6F* inPar, const MP3x3SF* hitErr, const MP3F* msP){//, MP3x3* inverse_temp, MP3x6* kGain, MP6x6SF* newErr){
  MP3x3 inverse_temp;
  MP3x6 kGain;
  MP6x6SF newErr;
  //MP6F newPar;
  KalmanGainInv(trkErr,hitErr,&inverse_temp);
  //__syncthreads(); 
  KalmanGain(trkErr,&inverse_temp,&kGain);
  //__syncthreads(); 
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    float xin = x(inPar,it);
    float yin = y(inPar,it);
    float zin = z(inPar,it);
    float ptin = 1./ipt(inPar,it);
    float phiin = phi(inPar,it);
    float thetain = theta(inPar,it);
    float xout = x(msP,it);
    float yout = y(msP,it);
    float zout = z(msP,it);
    float ydiff = y(msP,it) - y(inPar,it);
    float xnew = xin + (kGain.data[0*bsize+it]*(xout-xin)) +(kGain.data[1*bsize+it]*(yout-yin)); // removed "zout-zin" term since zin is set to zout thus the term is 0 anyway. 
    float ynew = yin + (kGain.data[3*bsize+it]*(xout-xin)) +(kGain.data[4*bsize+it]*(yout-yin));
    float znew = zin + (kGain.data[6*bsize+it]*(xout-xin)) +(kGain.data[7*bsize+it]*(yout-yin));
    float ptnew = ptin + (kGain.data[9*bsize+it]*(xout-xin)) +(kGain.data[10*bsize+it]*(yout-yin)); 
    float phinew = phiin + (kGain.data[12*bsize+it]*(xout-xin)) +(kGain.data[13*bsize+it]*(yout-yin));
    float thetanew = thetain + (kGain.data[15*bsize+it]*(xout-xin)) +(kGain.data[16*bsize+it]*(yout-yin)); 

    newErr.data[0*bsize+it] = trkErr->data[0*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[0*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[2*bsize+it]);
    newErr.data[1*bsize+it] = trkErr->data[1*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[2*bsize+it] = trkErr->data[2*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[3*bsize+it] = trkErr->data[3*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[4*bsize+it] = trkErr->data[4*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[5*bsize+it] = trkErr->data[5*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[6*bsize+it] = trkErr->data[6*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[7*bsize+it] = trkErr->data[7*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[8*bsize+it] = trkErr->data[8*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[9*bsize+it] = trkErr->data[9*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[10*bsize+it] = trkErr->data[10*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[11*bsize+it] = trkErr->data[11*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[12*bsize+it] = trkErr->data[12*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[13*bsize+it] = trkErr->data[13*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[14*bsize+it] = trkErr->data[14*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[15*bsize+it] = trkErr->data[15*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[16*bsize+it] = trkErr->data[16*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[17*bsize+it] = trkErr->data[17*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[18*bsize+it] = trkErr->data[18*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[19*bsize+it] = trkErr->data[19*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[14*bsize+it]);

    newErr.data[20*bsize+it] = trkErr->data[20*bsize+it] - (kGain.data[15*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[16*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[17*bsize+it]*trkErr->data[14*bsize+it]);

    setx(inPar,it,xnew );
    sety(inPar,it,ynew );
    setz(inPar,it,znew);
    setipt(inPar,it, ptnew);
    setphi(inPar,it, phinew);
    settheta(inPar,it, thetanew);
  }
  //__syncthreads(); 
  trkErr = &newErr;
}


__device__ __constant__ float kfact = 100/3.8;
__device__ __forceinline__ void propagateToZ(const MP6x6SF* inErr, const MP6F* inPar, const MP1I* inChg,const MP3F* msP, 
			  MP6x6SF* outErr, MP6F* outPar, struct MP6x6F* errorProp, struct MP6x6F* temp,const MP3x3SF* hitErr) {
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    const float zout = z(msP,it);
    const float k = q(inChg,it)*kfact;//100/3.8;
    const float deltaZ = zout - z(inPar,it);
    const float pt = 1./ipt(inPar,it);
    const float cosP = cosf(phi(inPar,it));
    const float sinP = sinf(phi(inPar,it));
    const float cosT = cosf(theta(inPar,it));
    const float sinT = sinf(theta(inPar,it));
    const float pxin = cosP*pt;
    const float pyin = sinP*pt;
    const float icosT = 1.0/cosT;
    const float icosTk = icosT/k;
    const float alpha = deltaZ*sinT*ipt(inPar,it)*icosTk;///(cosT*k);
    const float sina = sinf(alpha); // this can be approximated;
    const float cosa = cosf(alpha); // this can be approximated;
    setx(outPar,it, x(inPar,it) + k*(pxin*sina - pyin*(1.-cosa)) );
    sety(outPar,it, y(inPar,it) + k*(pyin*sina + pxin*(1.-cosa)) );
    setz(outPar,it,zout);
    setipt(outPar,it, ipt(inPar,it));
    setphi(outPar,it, phi(inPar,it)+alpha );
    settheta(outPar,it, theta(inPar,it) );
    
    const float sCosPsina = sinf(cosP*sina);
    const float cCosPsina = cosf(cosP*sina);
 
    for (size_t i=0;i<6;++i) errorProp->data[bsize*PosInMtrx(i,i,6) + it] = 1.;
    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k*pt)*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ*(icosT*icosTk);   
//    for (size_t i=0;i<6;++i) errorProp->data[bsize*PosInMtrx(i,i,6) + it] = 1.;
//    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)/cosT;
//    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*ipt(inPar,it))-k*(cosP*sina-sinP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
//    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k/ipt(inPar,it))*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
//    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*cosT);
//    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)/cosT;
//    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*ipt(inPar,it))-k*(sinP*sina+cosP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
//    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k/ipt(inPar,it))*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
//    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*cosT);
//    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT/(cosT*k);
//    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ/(cosT*k);
//    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ/(cosT*cosT*k);
  }
  //__syncthreads(); 
  MultHelixPropEndcap(errorProp, inErr, temp);
  //__syncthreads(); 
  MultHelixPropTranspEndcap(errorProp, temp, outErr);
}



__global__ void GPUsequence(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
  int ie_range;
  if(stream == num_streams){ ie_range = (int)(nevts%num_streams);}
  else{ie_range = (int)(nevts/num_streams);}
  for (size_t ie = blockIdx.x; ie<ie_range; ie+=gridDim.x){
    for(size_t ib = threadIdx.y; ib <nb; ib+=blockDim.y){
      const MPTRK* btracks = bTk(trk,ie,ib);
      MPTRK* obtracks = bTk(outtrk,ie,ib);
      for(int layer=0;layer<nlayer;++layer){	
        const MPHIT* bhits = bHit(hit,ie,ib,layer);
      /*__shared__*/ struct MP6x6F errorProp, temp; // using shared here causes a race hazard. idk why i did it this way, might be to include shared. maybe move to inside p2z function 
        propagateToZ(&(*btracks).cov, &(*btracks).par, &(*btracks).q, &(*bhits).pos, 
                     &(*obtracks).cov, &(*obtracks).par, &errorProp, &temp,&(*bhits).cov);
        KalmanUpdate(&(*obtracks).cov,&(*obtracks).par,&(*bhits).cov,&(*bhits).pos);
      }
    }
  }
}



int main (int argc, char* argv[]) {

  printf("RUNNING CUDA!!\n");
  printf("Streams: %d, blocks: %d, threads(x,y): (%d,%d)\n",num_streams,blockspergrid,threadsperblockx,threadsperblocky);
  ATRK inputtrk = {
     {-12.806846618652344, -7.723824977874756, 38.13014221191406,0.23732035065189902, -2.613372802734375, 0.35594117641448975},
     {6.290299552347278e-07,4.1375109560704004e-08,7.526661534029699e-07,2.0973730840978533e-07,1.5431574240665213e-07,9.626245400795597e-08,-2.804026640189443e-06,
      6.219111130687595e-06,2.649119409845118e-07,0.00253512163402557,-2.419662877381737e-07,4.3124190760040646e-07,3.1068903991780678e-09,0.000923913115050627,
      0.00040678296006807003,-7.755406890332818e-07,1.68539375883925e-06,6.676875566525437e-08,0.0008420574605423793,7.356584799406111e-05,0.0002306247719158348},
     1,
     {1, 0, 17, 16, 36, 35, 33, 34, 59, 58, 70, 85, 101, 102, 116, 117, 132, 133, 152, 169, 187, 202}
  };

  AHIT inputhit = {
     {-20.7824649810791, -12.24150276184082, 57.8067626953125},
     {2.545517190810642e-06,-2.6680759219743777e-06,2.8030024168401724e-06,0.00014160551654640585,0.00012282167153898627,11.385087966918945}
  };
  printf("track in pos: %f, %f, %f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2]);
  printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66(PosInMtrx(0,0,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(1,1,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(2,2,6))]);
  printf("hit in pos: %f %f %f \n", inputhit.pos[0], inputhit.pos[1], inputhit.pos[2]);

  printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
  printf("NITER=%d\n", NITER);
 
  long start_wall, end_wall, start_setup, end_setup; 
  struct timeval timecheck;
  hipEvent_t start, end, copy, copyback;
  hipEvent_t startcopy, copybackend;
  hipEventCreate(&start);
  hipEventCreate(&copy);
  hipEventCreate(&copyback);
  hipEventCreate(&startcopy);
  hipEventCreate(&copybackend);
  hipEventCreate(&end);
      
  gettimeofday(&timecheck, NULL);
  start_setup = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
  MPTRK* trk = prepareTracks(inputtrk);
  MPHIT* hit = prepareHits(inputhit);
  MPTRK* outtrk;
  hipMallocManaged((void**)&outtrk,nevts*nb*sizeof(MPTRK));
  dim3 grid(blockspergrid,1,1);
  dim3 block(threadsperblockx,threadsperblocky,1); 
  int device = -1;
  hipGetDevice(&device);
  int stream_chunk = ((int)(nevts/num_streams))*nb;//*sizeof(MPTRK);
  int stream_remainder = ((int)(nevts%num_streams))*nb;//*sizeof(MPTRK);
  int stream_range;
  if (stream_remainder == 0){ stream_range =num_streams;}
  else{stream_range = num_streams+1;}
  hipStream_t streams[stream_range];
  for (int s = 0; s<stream_range;s++){
    hipStreamCreate(&streams[s]);
    //hipStreamCreateWithFlags(&streams[s],hipStreamNonBlocking);
  }
  gettimeofday(&timecheck, NULL);
  end_setup = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
 

  printf("done preparing!\n");
  //long start, end;
  //long start2, end2;
  //struct timeval timecheck;

  printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*nb*sizeof(struct MPHIT));
  


  float elapsedtime,copytime,copybacktime,regiontime = 0;
  gettimeofday(&timecheck, NULL);
  start_wall = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
  hipEventRecord(start);	

  auto wall_start = std::chrono::high_resolution_clock::now();
  for(int itr=0; itr<NITER; itr++){
  hipEventRecord(startcopy);	
  hipEventSynchronize(startcopy);
  for (int s = 0; s<num_streams;s++){
    hipMemPrefetchAsync(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK), device,streams[s]);
      hipMemPrefetchAsync(hit+(s*stream_chunk*nlayer),nlayer*stream_chunk*sizeof(MPHIT), device,streams[s]);
    hipMemAdvise(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK),hipMemAdviseSetPreferredLocation,device);
      hipMemAdvise(hit+(s*stream_chunk*nlayer),nlayer*stream_chunk*sizeof(MPHIT),hipMemAdviseSetPreferredLocation,device);
    //}
    //hipStreamAttachMemAsync(streams[s],trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK),hipMemAttachHost);
    //hipStreamAttachMemAsync(streams[s],hit+(s*stream_chunk),stream_chunk*sizeof(MPHIT),hipMemAttachHost);
    //hipMemAdvise(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK),hipMemAdviseSetReadMostly,device);
    //hipMemAdvise(hit+(s*stream_chunk),stream_chunk*sizeof(MPHIT),hipMemAdviseSetReadMostly,device);
    //hipMemAdvise(trk+(s*stream_chunk),stream_chunk*sizeof(MPTRK),hipMemAdviseSetAccessedBy,device);
    //hipMemAdvise(hit+(s*stream_chunk),stream_chunk*sizeof(MPHIT),hipMemAdviseSetAccessedBy,device);
  }
  if(stream_remainder != 0){
    hipMemPrefetchAsync(trk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK), device,streams[num_streams]);
    hipMemAdvise(trk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK),hipMemAdviseSetPreferredLocation,device);
      hipMemPrefetchAsync(hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT), device,streams[num_streams]);
      hipMemAdvise(hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT),hipMemAdviseSetPreferredLocation,device);
  }
//  hipMemAdvise(trk,nevts*nb*sizeof(MPTRK),hipMemAdviseSetPreferredLocation,device);
//  hipMemAdvise(hit,nevts*nb*sizeof(MPHIT),hipMemAdviseSetPreferredLocation,device);
//  hipMemAdvise(trk,nevts*nb*sizeof(MPTRK),hipMemAdviseSetReadMostly,device);
//  hipMemAdvise(hit,nevts*nb*sizeof(MPHIT),hipMemAdviseSetReadMostly,device);


  hipEventRecord(copy);	
  hipEventSynchronize(copy);
    for (int s = 0; s<num_streams;s++){
  	  GPUsequence<<<grid,block,0,streams[s]>>>(trk+(s*stream_chunk),hit+(s*stream_chunk*nlayer),outtrk+(s*stream_chunk),s);
    }  
    if(stream_remainder != 0){
  	  GPUsequence<<<grid,block,0,streams[num_streams]>>>(trk+(num_streams*stream_chunk),hit+(num_streams*stream_chunk*nlayer),outtrk+(num_streams*stream_chunk),num_streams);
    }  
	  //hipDeviceSynchronize(); // Normal sync

  
  hipEventRecord(copyback);
  hipEventSynchronize(copyback);
    for (int s = 0; s<num_streams;s++){
      hipMemPrefetchAsync(outtrk+(s*stream_chunk),stream_chunk*sizeof(MPTRK), hipCpuDeviceId,streams[s]);
    }
    if(stream_remainder != 0){
      hipMemPrefetchAsync(outtrk+(num_streams*stream_chunk),stream_remainder*sizeof(MPTRK), hipCpuDeviceId,streams[num_streams]);
    }
  hipEventRecord(copybackend);	
  hipEventSynchronize(copybackend);
  float elapsedtime_itr,copytime_itr,copybacktime_itr = 0;
  hipEventElapsedTime(&elapsedtime_itr,copy,copyback);
  hipEventElapsedTime(&copytime_itr,startcopy,copy);
  hipEventElapsedTime(&copybacktime_itr,copyback,copybackend);
  elapsedtime += elapsedtime_itr;
  copytime += copytime_itr;
  copybacktime += copybacktime_itr;
  } //end itr loop
  hipDeviceSynchronize(); // shaves a few seconds
  auto wall_stop = std::chrono::high_resolution_clock::now();
  auto wall_diff = wall_stop - wall_start;
  auto wall_time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
  std::cout << "Wall clock time " << std::scientific << wall_time << " s" << std::endl;

  hipEventRecord(end);
  hipEventSynchronize(end);
  gettimeofday(&timecheck, NULL);
  end_wall = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
  //float elapsedtime,copytime,copybacktime,regiontime = 0;
  hipEventElapsedTime(&regiontime,start,end);
  //hipEventElapsedTime(&elapsedtime,copy,copyback);
  //hipEventElapsedTime(&copytime,start,copy);
  //hipEventElapsedTime(&copybacktime,copyback,end);
  
 
    for (int s = 0; s<stream_range;s++){
      hipStreamDestroy(streams[s]);
    }

   long walltime = end_wall-start_wall; 
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), (elapsedtime)*0.001, (elapsedtime)*0.001/(nevts*ntrks));
   printf("data region time=%f (s)\n", regiontime*0.001);
   printf("memory transfer time=%f (s) [%f, %f]\n", (copytime+copybacktime)*0.001, copytime*0.001, copybacktime*0.001);
   printf("setup time time=%f (s)\n", (end_setup-start_setup)*0.001);
   //printf("formatted %i %i %i %i %i %f %f %f %f %i\n",int(NITER),nevts,ntrks, bsize,nb, (0)*0.001, (regiontime)*0.001,  (0)*0.001, (end_setup-start_setup)*0.001, num_streams);
   printf("formatted %i %i %i %i %i %f %f %f %f %i\n",int(NITER),nevts,ntrks, bsize,nb, (elapsedtime)*0.001, (regiontime)*0.001,  (copytime+copybacktime)*0.001, (end_setup-start_setup)*0.001, num_streams);

   printf("wall region time=%f (s)\n", (end_wall-start_wall)*0.001);
   float avgx = 0, avgy = 0, avgz = 0;
   float avgpt = 0, avgphi = 0, avgtheta = 0;
   float avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
     }
   }
   avgpt = avgpt/float(nevts*ntrks);
   avgphi = avgphi/float(nevts*ntrks);
   avgtheta = avgtheta/float(nevts*ntrks);
   avgx = avgx/float(nevts*ntrks);
   avgy = avgy/float(nevts*ntrks);
   avgz = avgz/float(nevts*ntrks);
   avgdx = avgdx/float(nevts*ntrks);
   avgdy = avgdy/float(nevts*ntrks);
   avgdz = avgdz/float(nevts*ntrks);

   float stdx = 0, stdy = 0, stdz = 0;
   float stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
     }
   }

   stdx = sqrtf(stdx/float(nevts*ntrks));
   stdy = sqrtf(stdy/float(nevts*ntrks));
   stdz = sqrtf(stdz/float(nevts*ntrks));
   stddx = sqrtf(stddx/float(nevts*ntrks));
   stddy = sqrtf(stddy/float(nevts*ntrks));
   stddz = sqrtf(stddz/float(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);
	
   hipFree(trk);
   hipFree(hit);
   hipFree(outtrk);
   
return 0;
}

