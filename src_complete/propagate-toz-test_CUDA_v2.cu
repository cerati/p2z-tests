/*
icc propagate-toz-test.C -o propagate-toz-test.exe -fopenmp -O3
*/
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <iostream>

#include <chrono>
#include <iomanip>
#include <iostream>

#ifndef nevts
#define nevts 100
#endif
#ifndef bsize
#define bsize 128
#endif
#ifndef ntrks
#define ntrks 9600 //122880
#endif

#define nb    ntrks/bsize
#define smear 0.1

#ifndef NITER
#define NITER 5
#endif
#ifndef nlayer
#define nlayer 20
#endif
#ifndef num_stream
#define num_streams 10
#endif

#ifndef threadsperblockx
#define threadsperblockx 2
#endif
//#define threadsperblocky 1024/threadsperblockx
//#define threadsperblocky 512/threadsperblockx
#define threadsperblocky 32/threadsperblockx
#ifndef blockspergrid
#define blockspergrid 40
#endif

#define HOSTDEV __host__ __device__

HOSTDEV size_t PosInMtrx(size_t i, size_t j, size_t D) {
  return i*D+j;
}

HOSTDEV size_t SymOffsets33(size_t i) {
  const size_t offs[9] = {0, 1, 3, 1, 2, 4, 3, 4, 5};
  return offs[i];
}

HOSTDEV size_t SymOffsets66(size_t i) {
  const size_t offs[36] = {0, 1, 3, 6, 10, 15, 1, 2, 4, 7, 11, 16, 3, 4, 5, 8, 12, 17, 6, 7, 8, 9, 13, 18, 10, 11, 12, 13, 14, 19, 15, 16, 17, 18, 19, 20};
  return offs[i];
}


struct ATRK {
  float par[6];
  float cov[21];
  int q;
  int hitidx[22];
};

struct AHIT {
  float pos[3];
  float cov[6];
};

struct MP1I {
  int data[1*bsize];
};

struct MP22I {
  int data[22*bsize];
};

struct MP3F {
  float data[3*bsize];
};

struct MP6F {
  float data[6*bsize];
};

struct MP3x3 {
  float data[9*bsize];
};
struct MP3x6 {
  float data[18*bsize];
};

struct MP3x3SF {
  float data[6*bsize];
};

struct MP6x6SF {
  float data[21*bsize];
};

struct MP6x6F {
  float data[36*bsize];
};

struct MPTRK {
  MP6F    par;
  MP6x6SF cov;
  MP1I    q;
  MP22I   hitidx;
};

struct MPHIT {
  MP3F    pos;
  MP3x3SF cov;
};



float randn(float mu, float sigma) {
  float U1, U2, W, mult;
  static float X1, X2;
  static int call = 0;
  if (call == 1) {
    call = !call;
    return (mu + sigma * (float) X2);
  } do {
    U1 = -1 + ((float) rand () / RAND_MAX) * 2;
    U2 = -1 + ((float) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0);
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
  call = !call;
  return (mu + sigma * (float) X1);
}

MPTRK* prepareTracks(ATRK inputtrk) {
  //MPTRK* result = (MPTRK*) malloc(nevts*nb*sizeof(MPTRK));
  MPTRK* result;
  hipHostMalloc((void**)&result,nevts*nb*sizeof(MPTRK), hipHostMallocDefault);
  for (size_t ie=0;ie<nevts;++ie) {
    for (size_t ib=0;ib<nb;++ib) {
      for (size_t it=0;it<bsize;++it) {
        //par
        for (size_t ip=0;ip<6;++ip) {
          result[ib + nb*ie].par.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.par[ip];
        }
        //cov
        for (size_t ip=0;ip<21;++ip) {
          result[ib + nb*ie].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputtrk.cov[ip];
        }
        //q
        result[ib + nb*ie].q.data[it] = inputtrk.q-2*ceil(-0.5 + (float)rand() / RAND_MAX);//fixme check
      }
    }
  }
  return result;
}

MPHIT* prepareHits(AHIT inputhit) {
  //MPHIT* result = (MPHIT*) malloc(nlayer*nevts*nb*sizeof(MPHIT));
  MPHIT* result;
  hipHostMalloc((void**)&result,nlayer*nevts*nb*sizeof(MPHIT), hipHostMallocDefault);
  for (size_t lay=0;lay<nlayer;++lay) {
    for (size_t ie=0;ie<nevts;++ie) {
      for (size_t ib=0;ib<nb;++ib) {
        for (size_t it=0;it<bsize;++it) {
          //pos
          for (size_t ip=0;ip<3;++ip) {
            result[lay+nlayer*(ib + nb*ie)].pos.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.pos[ip];
          }
          //cov
          for (size_t ip=0;ip<6;++ip) {
            result[lay+nlayer*(ib + nb*ie)].cov.data[it + ip*bsize] = (1+smear*randn(0,1))*inputhit.cov[ip];
          }
        }
      }
    }
  }
  return result;
}


HOSTDEV MPTRK* bTk(MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}

HOSTDEV const MPTRK* bTk(const MPTRK* tracks, size_t ev, size_t ib) {
  return &(tracks[ib + nb*ev]);
}


HOSTDEV float q(const MP1I* bq, size_t it){
  return (*bq).data[it];
}

HOSTDEV float par(const MP6F* bpars, size_t it, size_t ipar){
  return (*bpars).data[it + ipar*bsize];
}
HOSTDEV float x    (const MP6F* bpars, size_t it){ return par(bpars, it, 0); }
HOSTDEV float y    (const MP6F* bpars, size_t it){ return par(bpars, it, 1); }
HOSTDEV float z    (const MP6F* bpars, size_t it){ return par(bpars, it, 2); }
HOSTDEV float ipt  (const MP6F* bpars, size_t it){ return par(bpars, it, 3); }
HOSTDEV float phi  (const MP6F* bpars, size_t it){ return par(bpars, it, 4); }
HOSTDEV float theta(const MP6F* bpars, size_t it){ return par(bpars, it, 5); }

HOSTDEV float par(const MPTRK* btracks, size_t it, size_t ipar){
  return par(&(*btracks).par,it,ipar);
}
HOSTDEV float x    (const MPTRK* btracks, size_t it){ return par(btracks, it, 0); }
HOSTDEV float y    (const MPTRK* btracks, size_t it){ return par(btracks, it, 1); }
HOSTDEV float z    (const MPTRK* btracks, size_t it){ return par(btracks, it, 2); }
HOSTDEV float ipt  (const MPTRK* btracks, size_t it){ return par(btracks, it, 3); }
HOSTDEV float phi  (const MPTRK* btracks, size_t it){ return par(btracks, it, 4); }
HOSTDEV float theta(const MPTRK* btracks, size_t it){ return par(btracks, it, 5); }

HOSTDEV float par(const MPTRK* tracks, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPTRK* btracks = bTk(tracks, ev, ib);
  size_t it = tk % bsize;
  return par(btracks, it, ipar);
}

HOSTDEV float x    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 0); }
HOSTDEV float y    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 1); }
HOSTDEV float z    (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 2); }
HOSTDEV float ipt  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 3); }
HOSTDEV float phi  (const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 4); }
HOSTDEV float theta(const MPTRK* tracks, size_t ev, size_t tk){ return par(tracks, ev, tk, 5); }

HOSTDEV void setpar(MP6F* bpars, size_t it, size_t ipar, float val){
  (*bpars).data[it + ipar*bsize] = val;
}
HOSTDEV void setx    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 0, val); }
HOSTDEV void sety    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 1, val); }
HOSTDEV void setz    (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 2, val); }
HOSTDEV void setipt  (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 3, val); }
HOSTDEV void setphi  (MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 4, val); }
HOSTDEV void settheta(MP6F* bpars, size_t it, float val){ return setpar(bpars, it, 5, val); }

HOSTDEV void setpar(MPTRK* btracks, size_t it, size_t ipar, float val){
  return setpar(&(*btracks).par,it,ipar,val);
}
HOSTDEV void setx    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 0, val); }
HOSTDEV void sety    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 1, val); }
HOSTDEV void setz    (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 2, val); }
HOSTDEV void setipt  (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 3, val); }
HOSTDEV void setphi  (MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 4, val); }
HOSTDEV void settheta(MPTRK* btracks, size_t it, float val){ return setpar(btracks, it, 5, val); }

HOSTDEV MPHIT* bHit(MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib) {
  return &(hits[ib + nb*ev]);
}
HOSTDEV const MPHIT* bHit(const MPHIT* hits, size_t ev, size_t ib,int lay) {
  return &(hits[lay + (ib*nlayer) +(ev*nlayer*nb)]);
}

HOSTDEV float pos(const MP3F* hpos, size_t it, size_t ipar){
  return (*hpos).data[it + ipar*bsize];
}
HOSTDEV float x(const MP3F* hpos, size_t it)    { return pos(hpos, it, 0); }
HOSTDEV float y(const MP3F* hpos, size_t it)    { return pos(hpos, it, 1); }
HOSTDEV float z(const MP3F* hpos, size_t it)    { return pos(hpos, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t it, size_t ipar){
  return pos(&(*hits).pos,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t it)    { return pos(hits, it, 0); }
HOSTDEV float y(const MPHIT* hits, size_t it)    { return pos(hits, it, 1); }
HOSTDEV float z(const MPHIT* hits, size_t it)    { return pos(hits, it, 2); }

HOSTDEV float pos(const MPHIT* hits, size_t ev, size_t tk, size_t ipar){
  size_t ib = tk/bsize;
  const MPHIT* bhits = bHit(hits, ev, ib);
  size_t it = tk % bsize;
  return pos(bhits,it,ipar);
}
HOSTDEV float x(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 0); }
HOSTDEV float y(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 1); }
HOSTDEV float z(const MPHIT* hits, size_t ev, size_t tk)    { return pos(hits, ev, tk, 2); }



#define N bsize
__forceinline__ __device__ void MultHelixPropEndcap(const MP6x6F* A, const MP6x6SF* B, MP6x6F* C) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = b[ 0*N+n] + a[ 2*N+n]*b[ 3*N+n] + a[ 3*N+n]*b[ 6*N+n] + a[ 4*N+n]*b[10*N+n] + a[ 5*N+n]*b[15*N+n];
    c[ 1*N+n] = b[ 1*N+n] + a[ 2*N+n]*b[ 4*N+n] + a[ 3*N+n]*b[ 7*N+n] + a[ 4*N+n]*b[11*N+n] + a[ 5*N+n]*b[16*N+n];
    c[ 2*N+n] = b[ 3*N+n] + a[ 2*N+n]*b[ 5*N+n] + a[ 3*N+n]*b[ 8*N+n] + a[ 4*N+n]*b[12*N+n] + a[ 5*N+n]*b[17*N+n];
    c[ 3*N+n] = b[ 6*N+n] + a[ 2*N+n]*b[ 8*N+n] + a[ 3*N+n]*b[ 9*N+n] + a[ 4*N+n]*b[13*N+n] + a[ 5*N+n]*b[18*N+n];
    c[ 4*N+n] = b[10*N+n] + a[ 2*N+n]*b[12*N+n] + a[ 3*N+n]*b[13*N+n] + a[ 4*N+n]*b[14*N+n] + a[ 5*N+n]*b[19*N+n];
    c[ 5*N+n] = b[15*N+n] + a[ 2*N+n]*b[17*N+n] + a[ 3*N+n]*b[18*N+n] + a[ 4*N+n]*b[19*N+n] + a[ 5*N+n]*b[20*N+n];
    c[ 6*N+n] = b[ 1*N+n] + a[ 8*N+n]*b[ 3*N+n] + a[ 9*N+n]*b[ 6*N+n] + a[10*N+n]*b[10*N+n] + a[11*N+n]*b[15*N+n];
    c[ 7*N+n] = b[ 2*N+n] + a[ 8*N+n]*b[ 4*N+n] + a[ 9*N+n]*b[ 7*N+n] + a[10*N+n]*b[11*N+n] + a[11*N+n]*b[16*N+n];
    c[ 8*N+n] = b[ 4*N+n] + a[ 8*N+n]*b[ 5*N+n] + a[ 9*N+n]*b[ 8*N+n] + a[10*N+n]*b[12*N+n] + a[11*N+n]*b[17*N+n];
    c[ 9*N+n] = b[ 7*N+n] + a[ 8*N+n]*b[ 8*N+n] + a[ 9*N+n]*b[ 9*N+n] + a[10*N+n]*b[13*N+n] + a[11*N+n]*b[18*N+n];
    c[10*N+n] = b[11*N+n] + a[ 8*N+n]*b[12*N+n] + a[ 9*N+n]*b[13*N+n] + a[10*N+n]*b[14*N+n] + a[11*N+n]*b[19*N+n];
    c[11*N+n] = b[16*N+n] + a[ 8*N+n]*b[17*N+n] + a[ 9*N+n]*b[18*N+n] + a[10*N+n]*b[19*N+n] + a[11*N+n]*b[20*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = 0;
    c[14*N+n] = 0;
    c[15*N+n] = 0;
    c[16*N+n] = 0;
    c[17*N+n] = 0;
    c[18*N+n] = b[ 6*N+n];
    c[19*N+n] = b[ 7*N+n];
    c[20*N+n] = b[ 8*N+n];
    c[21*N+n] = b[ 9*N+n];
    c[22*N+n] = b[13*N+n];
    c[23*N+n] = b[18*N+n];
    c[24*N+n] = a[26*N+n]*b[ 3*N+n] + a[27*N+n]*b[ 6*N+n] + b[10*N+n] + a[29*N+n]*b[15*N+n];
    c[25*N+n] = a[26*N+n]*b[ 4*N+n] + a[27*N+n]*b[ 7*N+n] + b[11*N+n] + a[29*N+n]*b[16*N+n];
    c[26*N+n] = a[26*N+n]*b[ 5*N+n] + a[27*N+n]*b[ 8*N+n] + b[12*N+n] + a[29*N+n]*b[17*N+n];
    c[27*N+n] = a[26*N+n]*b[ 8*N+n] + a[27*N+n]*b[ 9*N+n] + b[13*N+n] + a[29*N+n]*b[18*N+n];
    c[28*N+n] = a[26*N+n]*b[12*N+n] + a[27*N+n]*b[13*N+n] + b[14*N+n] + a[29*N+n]*b[19*N+n];
    c[29*N+n] = a[26*N+n]*b[17*N+n] + a[27*N+n]*b[18*N+n] + b[19*N+n] + a[29*N+n]*b[20*N+n];
    c[30*N+n] = b[15*N+n];
    c[31*N+n] = b[16*N+n];
    c[32*N+n] = b[17*N+n];
    c[33*N+n] = b[18*N+n];
    c[34*N+n] = b[19*N+n];
    c[35*N+n] = b[20*N+n];
  }
}

__forceinline__ __device__ void MultHelixPropTranspEndcap(MP6x6F* A, MP6x6F* B, MP6x6SF* C) {
  const float* a = A->data; //ASSUME_ALIGNED(a, 64);
  const float* b = B->data; //ASSUME_ALIGNED(b, 64);
  float* c = C->data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = b[ 0*N+n] + b[ 2*N+n]*a[ 2*N+n] + b[ 3*N+n]*a[ 3*N+n] + b[ 4*N+n]*a[ 4*N+n] + b[ 5*N+n]*a[ 5*N+n];
    c[ 1*N+n] = b[ 6*N+n] + b[ 8*N+n]*a[ 2*N+n] + b[ 9*N+n]*a[ 3*N+n] + b[10*N+n]*a[ 4*N+n] + b[11*N+n]*a[ 5*N+n];
    c[ 2*N+n] = b[ 7*N+n] + b[ 8*N+n]*a[ 8*N+n] + b[ 9*N+n]*a[ 9*N+n] + b[10*N+n]*a[10*N+n] + b[11*N+n]*a[11*N+n];
    c[ 3*N+n] = b[12*N+n] + b[14*N+n]*a[ 2*N+n] + b[15*N+n]*a[ 3*N+n] + b[16*N+n]*a[ 4*N+n] + b[17*N+n]*a[ 5*N+n];
    c[ 4*N+n] = b[13*N+n] + b[14*N+n]*a[ 8*N+n] + b[15*N+n]*a[ 9*N+n] + b[16*N+n]*a[10*N+n] + b[17*N+n]*a[11*N+n];
    c[ 5*N+n] = 0;
    c[ 6*N+n] = b[18*N+n] + b[20*N+n]*a[ 2*N+n] + b[21*N+n]*a[ 3*N+n] + b[22*N+n]*a[ 4*N+n] + b[23*N+n]*a[ 5*N+n];
    c[ 7*N+n] = b[19*N+n] + b[20*N+n]*a[ 8*N+n] + b[21*N+n]*a[ 9*N+n] + b[22*N+n]*a[10*N+n] + b[23*N+n]*a[11*N+n];
    c[ 8*N+n] = 0;
    c[ 9*N+n] = b[21*N+n];
    c[10*N+n] = b[24*N+n] + b[26*N+n]*a[ 2*N+n] + b[27*N+n]*a[ 3*N+n] + b[28*N+n]*a[ 4*N+n] + b[29*N+n]*a[ 5*N+n];
    c[11*N+n] = b[25*N+n] + b[26*N+n]*a[ 8*N+n] + b[27*N+n]*a[ 9*N+n] + b[28*N+n]*a[10*N+n] + b[29*N+n]*a[11*N+n];
    c[12*N+n] = 0;
    c[13*N+n] = b[27*N+n];
    c[14*N+n] = b[26*N+n]*a[26*N+n] + b[27*N+n]*a[27*N+n] + b[28*N+n] + b[29*N+n]*a[29*N+n];
    c[15*N+n] = b[30*N+n] + b[32*N+n]*a[ 2*N+n] + b[33*N+n]*a[ 3*N+n] + b[34*N+n]*a[ 4*N+n] + b[35*N+n]*a[ 5*N+n];
    c[16*N+n] = b[31*N+n] + b[32*N+n]*a[ 8*N+n] + b[33*N+n]*a[ 9*N+n] + b[34*N+n]*a[10*N+n] + b[35*N+n]*a[11*N+n];
    c[17*N+n] = 0;
    c[18*N+n] = b[33*N+n];
    c[19*N+n] = b[32*N+n]*a[26*N+n] + b[33*N+n]*a[27*N+n] + b[34*N+n] + b[35*N+n]*a[29*N+n];
    c[20*N+n] = b[35*N+n];
  }
}

__forceinline__ __device__ void KalmanGainInv(const MP6x6SF* A, const MP3x3SF* B, MP3x3* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the inverse of HpHt +R
  const float* a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const float* b = (*B).data; //ASSUME_ALIGNED(b, 64);
  float* c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    double det =
      ((a[0*N+n]+b[0*N+n])*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])))) -
      ((a[1*N+n]+b[1*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])))) +
      ((a[2*N+n]+b[2*N+n])*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n]))));
    double invdet = 1.0/det;

    c[ 0*N+n] =  invdet*(((a[ 6*N+n]+b[ 3*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 1*N+n] =  -1*invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 2*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[7*N+n]+b[4*N+n])));
    c[ 3*N+n] =  -1*invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[7*N+n]+b[4*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 4*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[11*N+n]+b[5*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[2*N+n]+b[2*N+n])));
    c[ 5*N+n] =  -1*invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 6*N+n] =  invdet*(((a[ 1*N+n]+b[ 1*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[6*N+n]+b[3*N+n])));
    c[ 7*N+n] =  -1*invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[7*N+n]+b[4*N+n])) - ((a[2*N+n]+b[2*N+n]) *(a[1*N+n]+b[1*N+n])));
    c[ 8*N+n] =  invdet*(((a[ 0*N+n]+b[ 0*N+n]) *(a[6*N+n]+b[3*N+n])) - ((a[1*N+n]+b[1*N+n]) *(a[1*N+n]+b[1*N+n])));
  }
  //__syncthreads(); 
}

__forceinline__ __device__ void KalmanGain(const MP6x6SF* A, const MP3x3* B, MP3x6* C) {
  // k = P Ht(HPHt + R)^-1
  // HpHt -> cov of x,y,z. take upper 3x3 matrix of P
  // This calculates the kalman gain 
  const float* a = (*A).data; //ASSUME_ALIGNED(a, 64);
  const float* b = (*B).data; //ASSUME_ALIGNED(b, 64);
  float* c = (*C).data;       //ASSUME_ALIGNED(c, 64);
  for(int n=threadIdx.x;n<N;n+=blockDim.x)
  {
    c[ 0*N+n] = a[0*N+n]*b[0*N+n] + a[1*N+n]*b[3*N+n] + a[2*N+n]*b[6*N+n];
    c[ 1*N+n] = a[0*N+n]*b[1*N+n] + a[1*N+n]*b[4*N+n] + a[2*N+n]*b[7*N+n];
    c[ 2*N+n] = a[0*N+n]*b[2*N+n] + a[1*N+n]*b[5*N+n] + a[2*N+n]*b[8*N+n];
    c[ 3*N+n] = a[1*N+n]*b[0*N+n] + a[6*N+n]*b[3*N+n] + a[7*N+n]*b[6*N+n];
    c[ 4*N+n] = a[1*N+n]*b[1*N+n] + a[6*N+n]*b[4*N+n] + a[7*N+n]*b[7*N+n];
    c[ 5*N+n] = a[1*N+n]*b[2*N+n] + a[6*N+n]*b[5*N+n] + a[7*N+n]*b[8*N+n];
    c[ 6*N+n] = a[2*N+n]*b[0*N+n] + a[7*N+n]*b[3*N+n] + a[11*N+n]*b[6*N+n];
    c[ 7*N+n] = a[2*N+n]*b[1*N+n] + a[7*N+n]*b[4*N+n] + a[11*N+n]*b[7*N+n];
    c[ 8*N+n] = a[2*N+n]*b[2*N+n] + a[7*N+n]*b[5*N+n] + a[11*N+n]*b[8*N+n];
    c[ 9*N+n] = a[3*N+n]*b[0*N+n] + a[8*N+n]*b[3*N+n] + a[12*N+n]*b[6*N+n];
    c[ 10*N+n] = a[3*N+n]*b[1*N+n] + a[8*N+n]*b[4*N+n] + a[12*N+n]*b[7*N+n];
    c[ 11*N+n] = a[3*N+n]*b[2*N+n] + a[8*N+n]*b[5*N+n] + a[12*N+n]*b[8*N+n];
    c[ 12*N+n] = a[4*N+n]*b[0*N+n] + a[9*N+n]*b[3*N+n] + a[13*N+n]*b[6*N+n];
    c[ 13*N+n] = a[4*N+n]*b[1*N+n] + a[9*N+n]*b[4*N+n] + a[13*N+n]*b[7*N+n];
    c[ 14*N+n] = a[4*N+n]*b[2*N+n] + a[9*N+n]*b[5*N+n] + a[13*N+n]*b[8*N+n];
    c[ 15*N+n] = a[5*N+n]*b[0*N+n] + a[10*N+n]*b[3*N+n] + a[14*N+n]*b[6*N+n];
    c[ 16*N+n] = a[5*N+n]*b[1*N+n] + a[10*N+n]*b[4*N+n] + a[14*N+n]*b[7*N+n];
    c[ 17*N+n] = a[5*N+n]*b[2*N+n] + a[10*N+n]*b[5*N+n] + a[14*N+n]*b[8*N+n];
  }
 // __syncthreads(); 
}

__forceinline__ __device__ void KalmanUpdate(MP6x6SF* trkErr, MP6F* inPar, const MP3x3SF* hitErr, const MP3F* msP){
  MP3x3 inverse_temp;
  MP3x6 kGain;
  MP6x6SF newErr;
 // __syncthreads(); 
  KalmanGainInv(trkErr,hitErr,&inverse_temp);
 // __syncthreads(); 
  KalmanGain(trkErr,&inverse_temp,&kGain);
 // __syncthreads(); 

  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    const float xin = x(inPar,it);
    const float yin = y(inPar,it);
    const float zin = z(inPar,it);
    const float ptin = 1./ipt(inPar,it); // is this pt or ipt? 
    const float phiin = phi(inPar,it);
    const float thetain = theta(inPar,it);
    const float xout = x(msP,it);
    const float yout = y(msP,it);
    const float zout = z(msP,it);
  
    float xnew = xin + (kGain.data[0*bsize+it]*(xout-xin)) +(kGain.data[1*bsize+it]*(yout-yin));
    float ynew = yin + (kGain.data[3*bsize+it]*(xout-xin)) +(kGain.data[4*bsize+it]*(yout-yin));
    float znew = zin + (kGain.data[6*bsize+it]*(xout-xin)) +(kGain.data[7*bsize+it]*(yout-yin));
    float ptnew = ptin + (kGain.data[9*bsize+it]*(xout-xin)) +(kGain.data[10*bsize+it]*(yout-yin));
    float phinew = phiin + (kGain.data[12*bsize+it]*(xout-xin)) +(kGain.data[13*bsize+it]*(yout-yin));
    float thetanew = thetain + (kGain.data[15*bsize+it]*(xout-xin)) +(kGain.data[16*bsize+it]*(yout-yin));
  
    newErr.data[0*bsize+it] = trkErr->data[0*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[0*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[2*bsize+it]);
    newErr.data[1*bsize+it] = trkErr->data[1*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[2*bsize+it] = trkErr->data[2*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[3*bsize+it] = trkErr->data[3*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[4*bsize+it] = trkErr->data[4*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[5*bsize+it] = trkErr->data[5*bsize+it] - (kGain.data[0*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[1*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[2*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[6*bsize+it] = trkErr->data[6*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[1*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[6*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[7*bsize+it]);
    newErr.data[7*bsize+it] = trkErr->data[7*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[8*bsize+it] = trkErr->data[8*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[9*bsize+it] = trkErr->data[9*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[10*bsize+it] = trkErr->data[10*bsize+it] - (kGain.data[3*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[4*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[5*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[11*bsize+it] = trkErr->data[11*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[2*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[7*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[11*bsize+it]);
    newErr.data[12*bsize+it] = trkErr->data[12*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[13*bsize+it] = trkErr->data[13*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[14*bsize+it] = trkErr->data[14*bsize+it] - (kGain.data[6*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[7*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[8*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[15*bsize+it] = trkErr->data[15*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[3*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[8*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[12*bsize+it]);
    newErr.data[16*bsize+it] = trkErr->data[16*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[17*bsize+it] = trkErr->data[17*bsize+it] - (kGain.data[9*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[10*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[11*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[18*bsize+it] = trkErr->data[18*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[4*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[9*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[13*bsize+it]);
    newErr.data[19*bsize+it] = trkErr->data[19*bsize+it] - (kGain.data[12*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[13*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[14*bsize+it]*trkErr->data[14*bsize+it]);
  
    newErr.data[20*bsize+it] = trkErr->data[20*bsize+it] - (kGain.data[15*bsize+it]*trkErr->data[5*bsize+it]+kGain.data[16*bsize+it]*trkErr->data[10*bsize+it]+kGain.data[17*bsize+it]*trkErr->data[14*bsize+it]);
  
    setx(inPar,it,xnew );
    sety(inPar,it,ynew );
    setz(inPar,it,znew);
    setipt(inPar,it, ptnew);
    setphi(inPar,it, phinew);
    settheta(inPar,it, thetanew);
  }
  __syncthreads(); 
  trkErr = &newErr;
}

__device__ __constant__ float kfact = 100/3.8;
__device__ __forceinline__ void propagateToZ(const MP6x6SF* inErr, const MP6F* inPar, const MP1I* inChg,const MP3F* msP, 
			  MP6x6SF* outErr, MP6F* outPar, struct MP6x6F* errorProp, struct MP6x6F* temp) {
        //struct MP6x6F* errorProp, temp; 
  for(size_t it=threadIdx.x;it<bsize;it+=blockDim.x){
    const float zout = z(msP,it);
    const float k = q(inChg,it)*kfact;//*100/3.8;
    const float deltaZ = zout - z(inPar,it);
    const float pt = 1./ipt(inPar,it);
    const float cosP = cosf(phi(inPar,it));
    const float sinP = sinf(phi(inPar,it));
    const float cosT = cosf(theta(inPar,it));
    const float sinT = sinf(theta(inPar,it));
    const float pxin = cosP*pt;
    const float pyin = sinP*pt;
    const float icosT = 1.0/cosT;
    const float icosTk = icosT/k;
    const float alpha = deltaZ*sinT*ipt(inPar,it)*icosTk;
    //const float alpha = deltaZ*sinT*ipt(inPar,it)/(cosT*k);
    const float sina = sinf(alpha); // this can be approximated;
    const float cosa = cosf(alpha); // this can be approximated;
    setx(outPar,it, x(inPar,it) + k*(pxin*sina - pyin*(1.-cosa)) );
    sety(outPar,it, y(inPar,it) + k*(pyin*sina + pxin*(1.-cosa)) );
    setz(outPar,it,zout);
    setipt(outPar,it, ipt(inPar,it));
    setphi(outPar,it, phi(inPar,it)+alpha );
    settheta(outPar,it, theta(inPar,it) );
    
    const float sCosPsina = sinf(cosP*sina);
    const float cCosPsina = cosf(cosP*sina);
    
    for (size_t i=0;i<6;++i) errorProp->data[bsize*PosInMtrx(i,i,6) + it] = 1.;
    errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)*icosT;
    errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*pt)-k*(cosP*sina-sinP*(1.-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k*pt)*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
    errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)*icosT;
    errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*pt)-k*(sinP*sina+cosP*(1.-cCosPsina))*(pt*pt);
    errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k*pt)*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)*(icosT*icosT);
    errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ*(icosTk);
    errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ*(icosT*icosTk);
    //errorProp->data[bsize*PosInMtrx(0,2,6) + it] = cosP*sinT*(sinP*cosa*sCosPsina-cosa)/cosT;
    //errorProp->data[bsize*PosInMtrx(0,3,6) + it] = cosP*sinT*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*ipt(inPar,it))-k*(cosP*sina-sinP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
    //errorProp->data[bsize*PosInMtrx(0,4,6) + it] = (k/ipt(inPar,it))*(-sinP*sina+sinP*sinP*sina*sCosPsina-cosP*(1.-cCosPsina));
    //errorProp->data[bsize*PosInMtrx(0,5,6) + it] = cosP*deltaZ*cosa*(1.-sinP*sCosPsina)/(cosT*cosT);
    //errorProp->data[bsize*PosInMtrx(1,2,6) + it] = cosa*sinT*(cosP*cosP*sCosPsina-sinP)/cosT;
    //errorProp->data[bsize*PosInMtrx(1,3,6) + it] = sinT*deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*ipt(inPar,it))-k*(sinP*sina+cosP*(1.-cCosPsina))/(ipt(inPar,it)*ipt(inPar,it));
    //errorProp->data[bsize*PosInMtrx(1,4,6) + it] = (k/ipt(inPar,it))*(-sinP*(1.-cCosPsina)-sinP*cosP*sina*sCosPsina+cosP*sina);
    //errorProp->data[bsize*PosInMtrx(1,5,6) + it] = deltaZ*cosa*(cosP*cosP*sCosPsina+sinP)/(cosT*cosT);
    //errorProp->data[bsize*PosInMtrx(4,2,6) + it] = -ipt(inPar,it)*sinT/(cosT*k);
    //errorProp->data[bsize*PosInMtrx(4,3,6) + it] = sinT*deltaZ/(cosT*k);
    //errorProp->data[bsize*PosInMtrx(4,5,6) + it] = ipt(inPar,it)*deltaZ/(cosT*cosT*k);
  }
  __syncthreads(); 
  MultHelixPropEndcap(errorProp, inErr, temp);
  __syncthreads(); 
  MultHelixPropTranspEndcap(errorProp, temp, outErr);
}



__device__ __constant__ int ie_range = (int) nevts/num_streams; 
__device__ __constant__ int ie_rangeR = (int) nevts%num_streams; 
__global__ void GPUsequence(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
   //__shared__ int ie_range;
   //ie_range = (int)(nevts/num_streams);
  //if(stream == num_streams){ ie_range = (int)(nevts%num_streams);}
  //else{ie_range = (int)(nevts/num_streams);}
        /*__shared__*/ struct MP6x6F errorProp, temp; // shared memory here causes a race condition. Probably move to inside the p2z function? i forgot why I did it this way to begin with. maybe to make it shared?
  for (size_t ie = blockIdx.x; ie<ie_range; ie+=gridDim.x){
    for(size_t ib = threadIdx.y; ib <nb; ib+=blockDim.y){
      const MPTRK* btracks = bTk(trk,ie,ib);
      MPTRK* obtracks = bTk(outtrk,ie,ib);
      for (int layer=0;layer<nlayer;++layer){	
        const MPHIT* bhits = bHit(hit,ie,ib,layer);
        propagateToZ(&(*btracks).cov, &(*btracks).par, &(*btracks).q, &(*bhits).pos, 
                     &(*obtracks).cov, &(*obtracks).par, &errorProp, &temp);
 //       __syncthreads();
        KalmanUpdate(&(*obtracks).cov,&(*obtracks).par,&(*bhits).cov,&(*bhits).pos);
      }
    }
  }
}
__global__ void GPUsequenceR(MPTRK* trk, MPHIT* hit, MPTRK* outtrk, const int stream){
  //const int ie_range = (int)(nevts%num_streams);
  //if(stream == num_streams){ ie_range = (int)(nevts%num_streams);}
  //else{ie_range = (int)(nevts/num_streams);}
  for (size_t ie = blockIdx.x; ie<ie_rangeR; ie+=gridDim.x){
    for(size_t ib = threadIdx.y; ib <nb; ib+=blockDim.y){
      const MPTRK* btracks = bTk(trk,ie,ib);
      MPTRK* obtracks = bTk(outtrk,ie,ib);
      for (int layer=0;layer<nlayer;++layer){	
        const MPHIT* bhits = bHit(hit,ie,ib,layer);
        /*__shared__*/ struct MP6x6F errorProp, temp; // shared memory here causes a race condition. Probably move to inside the p2z function? i forgot why I did it this way to begin with. maybe to make it shared?
        propagateToZ(&(*btracks).cov, &(*btracks).par, &(*btracks).q, &(*bhits).pos, 
                     &(*obtracks).cov, &(*obtracks).par, &errorProp, &temp);
       // __syncthreads();
        KalmanUpdate(&(*obtracks).cov,&(*obtracks).par,&(*bhits).cov,&(*bhits).pos);
      }
    }
  }
}
inline void transferAsyncTrk(MPTRK* trk, MPTRK* trk_dev, hipStream_t stream){

  hipMemcpyAsync(trk_dev, trk, nevts*nb*sizeof(MPTRK), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(&trk_dev->par, &trk->par, sizeof(MP6F), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(&((trk_dev->par).data), &((trk->par).data), 6*bsize*sizeof(float), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(&trk_dev->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(&((trk_dev->cov).data), &((trk->cov).data), 36*bsize*sizeof(float), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(&trk_dev->q, &trk->q, sizeof(MP1I), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(&((trk_dev->q).data), &((trk->q).data), 1*bsize*sizeof(int), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(&trk_dev->hitidx, &trk->hitidx, sizeof(MP22I), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(&((trk_dev->hitidx).data), &((trk->hitidx).data), 22*bsize*sizeof(int), hipMemcpyHostToDevice, stream);
  
}
inline void transferAsyncHit(MPHIT* hit, MPHIT* hit_dev, hipStream_t stream){

    hipMemcpyAsync(hit_dev,hit,nlayer*nevts*nb*sizeof(MPHIT), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(&hit_dev->pos,&hit->pos,sizeof(MP3F), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(&(hit_dev->pos).data,&(hit->pos).data,3*bsize*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(&hit_dev->cov,&hit->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(&(hit_dev->cov).data,&(hit->cov).data,6*bsize*sizeof(float), hipMemcpyHostToDevice, stream);
}
inline void transfer_backAsync(MPTRK* trk, MPTRK* trk_host,hipStream_t stream){
  hipMemcpyAsync(trk_host, trk, nevts*nb*sizeof(MPTRK), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&trk_host->par, &trk->par, sizeof(MP6F), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&((trk_host->par).data), &((trk->par).data), 6*bsize*sizeof(float), hipMemcpyDeviceToHost,stream);
  hipMemcpyAsync(&trk_host->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&((trk_host->cov).data), &((trk->cov).data), 36*bsize*sizeof(float), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&trk_host->q, &trk->q, sizeof(MP1I), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&((trk_host->q).data), &((trk->q).data), 1*bsize*sizeof(int), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&trk_host->hitidx, &trk->hitidx, sizeof(MP22I), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&((trk_host->hitidx).data), &((trk->hitidx).data), 22*bsize*sizeof(int), hipMemcpyDeviceToHost,stream);
}
inline void transfer(MPTRK* trk, MPHIT* hit, MPTRK* trk_dev, MPHIT* hit_dev){

  hipMemcpy(trk_dev, trk, nevts*nb*sizeof(MPTRK), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->par, &trk->par, sizeof(MP6F), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->par).data), &((trk->par).data), 6*bsize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->cov).data), &((trk->cov).data), 36*bsize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->q, &trk->q, sizeof(MP1I), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->q).data), &((trk->q).data), 1*bsize*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&trk_dev->hitidx, &trk->hitidx, sizeof(MP22I), hipMemcpyHostToDevice);
  hipMemcpy(&((trk_dev->hitidx).data), &((trk->hitidx).data), 22*bsize*sizeof(int), hipMemcpyHostToDevice);
  
  hipMemcpy(hit_dev,hit,nevts*nb*sizeof(MPHIT), hipMemcpyHostToDevice);
  hipMemcpy(&hit_dev->pos,&hit->pos,sizeof(MP3F), hipMemcpyHostToDevice);
  hipMemcpy(&(hit_dev->pos).data,&(hit->pos).data,3*bsize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&hit_dev->cov,&hit->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice);
  hipMemcpy(&(hit_dev->cov).data,&(hit->cov).data,6*bsize*sizeof(float), hipMemcpyHostToDevice);
}
inline void transfer_back(MPTRK* trk, MPTRK* trk_host){
  hipMemcpy(trk_host, trk, nevts*nb*sizeof(MPTRK), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->par, &trk->par, sizeof(MP6F), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->par).data), &((trk->par).data), 6*bsize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->cov, &trk->cov, sizeof(MP6x6SF), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->cov).data), &((trk->cov).data), 36*bsize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->q, &trk->q, sizeof(MP1I), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->q).data), &((trk->q).data), 1*bsize*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&trk_host->hitidx, &trk->hitidx, sizeof(MP22I), hipMemcpyDeviceToHost);
  hipMemcpy(&((trk_host->hitidx).data), &((trk->hitidx).data), 22*bsize*sizeof(int), hipMemcpyDeviceToHost);
}

int main (int argc, char* argv[]) {

  printf("RUNNING CUDA!!\n");
  printf("Streams: %d, blocks: %d, threads(x,y): (%d,%d)\n",num_streams,blockspergrid,threadsperblockx,threadsperblocky);
  int itr;
  ATRK inputtrk = {
     {-12.806846618652344, -7.723824977874756, 38.13014221191406,0.23732035065189902, -2.613372802734375, 0.35594117641448975},
     {6.290299552347278e-07,4.1375109560704004e-08,7.526661534029699e-07,2.0973730840978533e-07,1.5431574240665213e-07,9.626245400795597e-08,-2.804026640189443e-06,
      6.219111130687595e-06,2.649119409845118e-07,0.00253512163402557,-2.419662877381737e-07,4.3124190760040646e-07,3.1068903991780678e-09,0.000923913115050627,
      0.00040678296006807003,-7.755406890332818e-07,1.68539375883925e-06,6.676875566525437e-08,0.0008420574605423793,7.356584799406111e-05,0.0002306247719158348},
     1,
     {1, 0, 17, 16, 36, 35, 33, 34, 59, 58, 70, 85, 101, 102, 116, 117, 132, 133, 152, 169, 187, 202}
  };

  AHIT inputhit = {
     {-20.7824649810791, -12.24150276184082, 57.8067626953125},
     {2.545517190810642e-06,-2.6680759219743777e-06,2.8030024168401724e-06,0.00014160551654640585,0.00012282167153898627,11.385087966918945}
  };
  printf("track in pos: %f, %f, %f \n", inputtrk.par[0], inputtrk.par[1], inputtrk.par[2]);
  printf("track in cov: %.2e, %.2e, %.2e \n", inputtrk.cov[SymOffsets66(PosInMtrx(0,0,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(1,1,6))],
                                              inputtrk.cov[SymOffsets66(PosInMtrx(2,2,6))]);
  printf("hit in pos: %f %f %f \n", inputhit.pos[0], inputhit.pos[1], inputhit.pos[2]);

  printf("produce nevts=%i ntrks=%i smearing by=%f \n", nevts, ntrks, smear);
  printf("NITER=%d\n", NITER);
 
  long start_wall, end_wall, start_setup, end_setup; 
  struct timeval timecheck;
  hipEvent_t start, end, copy, copyback;
  hipEvent_t startcopy, copybackend;
  hipEventCreate(&start);
  hipEventCreate(&copy);
  hipEventCreate(&copyback);
  hipEventCreate(&startcopy);
  hipEventCreate(&copybackend);
  hipEventCreate(&end);
      
  gettimeofday(&timecheck, NULL);
  start_setup = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
//  cudaDeviceSetCacheConfig(cudaFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(GPUsequence),hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(GPUsequenceR),hipFuncCachePreferL1);
  MPTRK* trk = prepareTracks(inputtrk);
  MPHIT* hit = prepareHits(inputhit);
  //cudaHostRegister((void**)&trk,nevts*nb*sizeof(MPTRK),cudaHostRegisterDefault);
  //cudaHostRegister((void**)&hit,nlayer*nevts*nb*sizeof(MPHIT),cudaHostRegisterDefault);
  MPTRK* trk_dev;
  MPHIT* hit_dev;
  //MPTRK* outtrk= (MPTRK*) malloc(nevts*nb*sizeof(MPTRK)); 
  MPTRK* outtrk;
  hipHostMalloc((void**)&outtrk,nevts*nb*sizeof(MPTRK), hipHostMallocDefault); 
  MPTRK* outtrk_dev;
  hipMalloc((MPTRK**)&trk_dev,nevts*nb*sizeof(MPTRK));
  hipMalloc((MPHIT**)&hit_dev,nlayer*nevts*nb*sizeof(MPHIT));
  hipMalloc((MPTRK**)&outtrk_dev,nevts*nb*sizeof(MPTRK));
  dim3 grid(blockspergrid,1,1);
  dim3 block(threadsperblockx,threadsperblocky,1); 
  int device = -1;
  hipGetDevice(&device);
  int stream_chunk = ((int)(nevts/num_streams))*nb;//*sizeof(MPTRK);
  int stream_remainder = ((int)(nevts%num_streams))*nb;//*sizeof(MPTRK);
  int stream_range;
  if (stream_remainder == 0){ stream_range =num_streams;}
  else{stream_range = num_streams+1;}
  hipStream_t streams[stream_range];
  for (int s = 0; s<stream_range;s++){
    //cudaStreamCreateWithFlags(&streams[s],cudaStreamNonBlocking);
    hipStreamCreate(&streams[s]);
  }
  gettimeofday(&timecheck, NULL);
  end_setup = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
 

  printf("done preparing!\n");
  //long start, end;
  //long start2, end2;
  //struct timeval timecheck;

  printf("Size of struct MPTRK trk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct MPTRK outtrk[] = %ld\n", nevts*nb*sizeof(struct MPTRK));
  printf("Size of struct struct MPHIT hit[] = %ld\n", nevts*nb*sizeof(struct MPHIT));
  


  float elapsedtime,copytime,copybacktime,regiontime = 0;
  hipEventRecord(start);	
  hipEventSynchronize(start);
  //gettimeofday(&timecheck, NULL);
 // start_wall = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;


  auto wall_start = std::chrono::high_resolution_clock::now();
  for(itr=0; itr<NITER; itr++){
    //  transfer(trk,hit, trk_dev,hit_dev);
  //cudaEventRecord(startcopy);	
  //cudaEventSynchronize(startcopy);
    for (int s = 0; s<num_streams;s++){
//      transferAsyncTrk(trk, trk_dev,streams[s]);
      hipMemcpyAsync(trk_dev+(s*stream_chunk), trk+(s*stream_chunk), stream_chunk*sizeof(MPTRK), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(trk_dev+(s*stream_chunk))->par, &(trk+(s*stream_chunk))->par, sizeof(MP6F), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(((trk_dev+(s*stream_chunk))->par).data), &(((trk+(s*stream_chunk))->par).data), 6*bsize*sizeof(float), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(trk_dev+(s*stream_chunk))->cov, &(trk+(s*stream_chunk))->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(((trk_dev+(s*stream_chunk))->cov).data), &(((trk+(s*stream_chunk))->cov).data), 36*bsize*sizeof(float), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(trk_dev+(s*stream_chunk))->q, &(trk+(s*stream_chunk))->q, sizeof(MP1I), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(((trk_dev+(s*stream_chunk))->q).data), &(((trk+(s*stream_chunk))->q).data), 1*bsize*sizeof(int), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(trk_dev+(s*stream_chunk))->hitidx, &(trk+(s*stream_chunk))->hitidx, sizeof(MP22I), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(((trk_dev+(s*stream_chunk))->hitidx).data), &(((trk+(s*stream_chunk))->hitidx).data), 22*bsize*sizeof(int), hipMemcpyHostToDevice, streams[s]);
      
      hipMemcpyAsync(hit_dev+(s*stream_chunk*nlayer),hit+(s*stream_chunk),nlayer*stream_chunk*sizeof(MPHIT), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(hit_dev+(s*stream_chunk*nlayer))->pos,&(hit+(s*stream_chunk*nlayer))->pos,sizeof(MP3F), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&((hit_dev+(s*stream_chunk*nlayer))->pos).data,&((hit+(s*stream_chunk*nlayer))->pos).data,3*bsize*sizeof(float), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&(hit_dev+(s*stream_chunk*nlayer))->cov,&(hit+(s*stream_chunk*nlayer))->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice, streams[s]);
      hipMemcpyAsync(&((hit_dev+(s*stream_chunk*nlayer))->cov).data,&((hit+(s*stream_chunk*nlayer))->cov).data,6*bsize*sizeof(float), hipMemcpyHostToDevice, streams[s]);
    }  
    if(stream_remainder != 0){
      hipMemcpyAsync(trk_dev+(num_streams*stream_chunk), trk+(num_streams*stream_chunk), stream_remainder*sizeof(MPTRK), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(trk_dev+(num_streams*stream_chunk))->par, &(trk+(num_streams*stream_chunk))->par, sizeof(MP6F), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(((trk_dev+(num_streams*stream_chunk))->par).data), &(((trk+(num_streams*stream_chunk))->par).data), 6*bsize*sizeof(float), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(trk_dev+(num_streams*stream_chunk))->cov, &(trk+(num_streams*stream_chunk))->cov, sizeof(MP6x6SF), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(((trk_dev+(num_streams*stream_chunk))->cov).data), &(((trk+(num_streams*stream_chunk))->cov).data), 36*bsize*sizeof(float), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(trk_dev+(num_streams*stream_chunk))->q, &(trk+(num_streams*stream_chunk))->q, sizeof(MP1I), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(((trk_dev+(num_streams*stream_chunk))->q).data), &(((trk+(num_streams*stream_chunk))->q).data), 1*bsize*sizeof(int), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(trk_dev+(num_streams*stream_chunk))->hitidx, &(trk+(num_streams*stream_chunk))->hitidx, sizeof(MP22I), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(((trk_dev+(num_streams*stream_chunk))->hitidx).data), &(((trk+(num_streams*stream_chunk))->hitidx).data), 22*bsize*sizeof(int), hipMemcpyHostToDevice, streams[num_streams]);
      
      hipMemcpyAsync(hit_dev+(num_streams*stream_chunk*nlayer),hit+(num_streams*stream_chunk*nlayer),nlayer*stream_remainder*sizeof(MPHIT), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(hit_dev+(num_streams*stream_chunk*nlayer))->pos,&(hit+(num_streams*stream_chunk*nlayer))->pos,sizeof(MP3F), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&((hit_dev+(num_streams*stream_chunk*nlayer))->pos).data,&((hit+(num_streams*stream_chunk*nlayer))->pos).data,3*bsize*sizeof(float), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&(hit_dev+(num_streams*stream_chunk*nlayer))->cov,&(hit+(num_streams*stream_chunk*nlayer))->cov,sizeof(MP3x3SF), hipMemcpyHostToDevice, streams[num_streams]);
      hipMemcpyAsync(&((hit_dev+(num_streams*stream_chunk*nlayer))->cov).data,&((hit+(num_streams*stream_chunk*nlayer))->cov).data,6*bsize*sizeof(float), hipMemcpyHostToDevice, streams[num_streams]);
    }

  //cudaEventRecord(copy);	
  //cudaEventSynchronize(copy);
	  //cudaDeviceSynchronize(); 
    for (int s = 0; s<num_streams;++s){
  	  GPUsequence<<<grid,block,0,streams[s]>>>(trk_dev+(s*stream_chunk),hit_dev+(s*stream_chunk*nlayer),outtrk_dev+(s*stream_chunk),s);
    }  
    if(stream_remainder != 0){
  	  GPUsequenceR<<<grid,block,0,streams[num_streams]>>>(trk_dev+(num_streams*stream_chunk),hit_dev+(num_streams*stream_chunk*nlayer),outtrk_dev+(num_streams*stream_chunk),num_streams);
    }
	  //cudaDeviceSynchronize(); 
//     // transfer_back(outtrk_dev,outtrk); 
  //cudaEventRecord(copyback);	
  //cudaEventSynchronize(copyback);
    for (int s = 0; s<num_streams;s++){
      hipMemcpyAsync(outtrk+(s*stream_chunk), outtrk_dev+(s*stream_chunk), stream_chunk*sizeof(MPTRK), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(outtrk+(s*stream_chunk))->par, &(outtrk_dev+(s*stream_chunk))->par, sizeof(MP6F), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(((outtrk+(s*stream_chunk))->par).data), &(((outtrk_dev+(s*stream_chunk))->par).data), 6*bsize*sizeof(float), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(outtrk+(s*stream_chunk))->cov, &(outtrk_dev+(s*stream_chunk))->cov, sizeof(MP6x6SF), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(((outtrk+(s*stream_chunk))->cov).data), &(((outtrk_dev+(s*stream_chunk))->cov).data), 36*bsize*sizeof(float), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(outtrk+(s*stream_chunk))->q, &(outtrk_dev+(s*stream_chunk))->q, sizeof(MP1I), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(((outtrk+(s*stream_chunk))->q).data), &(((outtrk_dev+(s*stream_chunk))->q).data), 1*bsize*sizeof(int), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(outtrk+(s*stream_chunk))->hitidx, &(outtrk_dev+(s*stream_chunk))->hitidx, sizeof(MP22I), hipMemcpyDeviceToHost, streams[s]);
      hipMemcpyAsync(&(((outtrk+(s*stream_chunk))->hitidx).data), &(((outtrk_dev+(s*stream_chunk))->hitidx).data), 22*bsize*sizeof(int), hipMemcpyDeviceToHost, streams[s]);
    }
    if(stream_remainder != 0){
      hipMemcpyAsync(outtrk+(num_streams*stream_chunk), outtrk_dev+(num_streams*stream_chunk), stream_remainder*sizeof(MPTRK), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(outtrk+(num_streams*stream_chunk))->par, &(outtrk_dev+(num_streams*stream_chunk))->par, sizeof(MP6F), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(((outtrk+(num_streams*stream_chunk))->par).data), &(((outtrk_dev+(num_streams*stream_chunk))->par).data), 6*bsize*sizeof(float), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(outtrk+(num_streams*stream_chunk))->cov, &(outtrk_dev+(num_streams*stream_chunk))->cov, sizeof(MP6x6SF), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(((outtrk+(num_streams*stream_chunk))->cov).data), &(((outtrk_dev+(num_streams*stream_chunk))->cov).data), 36*bsize*sizeof(float), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(outtrk+(num_streams*stream_chunk))->q, &(outtrk_dev+(num_streams*stream_chunk))->q, sizeof(MP1I), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(((outtrk+(num_streams*stream_chunk))->q).data), &(((outtrk_dev+(num_streams*stream_chunk))->q).data), 1*bsize*sizeof(int), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(outtrk+(num_streams*stream_chunk))->hitidx, &(outtrk_dev+(num_streams*stream_chunk))->hitidx, sizeof(MP22I), hipMemcpyDeviceToHost, streams[num_streams]);
      hipMemcpyAsync(&(((outtrk+(num_streams*stream_chunk))->hitidx).data), &(((outtrk_dev+(num_streams*stream_chunk))->hitidx).data), 22*bsize*sizeof(int), hipMemcpyDeviceToHost, streams[num_streams]);
    }

  //cudaEventRecord(copybackend);	
  //cudaEventSynchronize(copybackend);
  //float elapsedtime_itr,copytime_itr,copybacktime_itr = 0;
  //cudaEventElapsedTime(&elapsedtime_itr,copy,copyback);
  //cudaEventElapsedTime(&copytime_itr,startcopy,copy);
  //cudaEventElapsedTime(&copybacktime_itr,copyback,copybackend);
  //elapsedtime += elapsedtime_itr; 
  //copytime += copytime_itr;
  //copybacktime += copybacktime_itr;
  //printf("copy %f\n",copytime);
	  //cudaDeviceSynchronize(); // Normal sync

  } //end itr loop
  
  hipDeviceSynchronize(); 
  auto wall_stop = std::chrono::high_resolution_clock::now();
  auto wall_diff = wall_stop - wall_start;
  auto wall_time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(wall_diff).count()) / 1e6;
  std::cout << "Wall clock time " << std::scientific << wall_time << " s" << std::endl;

//  gettimeofday(&timecheck, NULL);
//  end_wall = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
  hipEventRecord(end);
  hipEventSynchronize(end);
  //float elapsedtime,copytime,copybacktime,regiontime = 0;
  hipEventElapsedTime(&regiontime,start,end);
  //cudaEventElapsedTime(&elapsedtime,copy,copyback);
  //cudaEventElapsedTime(&copytime,start,copy);
  //cudaEventElapsedTime(&copybacktime,copyback,end);
 

  for (int s = 0; s<stream_range;s++){
    hipStreamDestroy(streams[s]);
  }
 

 //  long walltime = end_wall-start_wall; 
   printf("done ntracks=%i tot time=%f (s) time/trk=%e (s)\n", nevts*ntrks*int(NITER), (elapsedtime)*0.001, (elapsedtime)*0.001/(nevts*ntrks));
   printf("data region time=%f (s)\n", regiontime*0.001);
   printf("memory transfer time=%f (s) [%f,%f]\n", (copytime+copybacktime)*0.001,copytime*0.001,copybacktime*0.001);
   printf("setup time time=%f (s)\n", (end_setup-start_setup)*0.001);
   printf("formatted %i %i %i %i %i %f %f %f %f %i\n",int(NITER),nevts,ntrks,bsize, nb, (elapsedtime)*0.001, (regiontime)*0.001,  (copytime+copybacktime)*0.001, (end_setup-start_setup)*0.001, num_streams);

 //  printf("wall region time=%f (s)\n", (walltime)*0.001);
   float avgx = 0, avgy = 0, avgz = 0;
   float avgpt = 0, avgphi = 0, avgtheta = 0;
   float avgdx = 0, avgdy = 0, avgdz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       float pt_ = 1./ipt(outtrk,ie,it);
       float phi_ = phi(outtrk,ie,it);
       float theta_ = theta(outtrk,ie,it);
       //if(x_ ==0 || y_==0||z_==0){
       //printf("x: %f,y: %f,z: %f, ie: %d, it: %f\n",x_,y_,z_,ie,it);
       //continue;
       //}
       avgpt += pt_;
       avgphi += phi_;
       avgtheta += theta_;
       avgx += x_;
       avgy += y_;
       avgz += z_;
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       //if(x_ ==0 || y_==0 || z_==0){continue;}
       avgdx += (x_-hx_)/x_;
       avgdy += (y_-hy_)/y_;
       avgdz += (z_-hz_)/z_;
     }
   }
   avgpt = avgpt/float(nevts*ntrks);
   avgphi = avgphi/float(nevts*ntrks);
   avgtheta = avgtheta/float(nevts*ntrks);
   avgx = avgx/float(nevts*ntrks);
   avgy = avgy/float(nevts*ntrks);
   avgz = avgz/float(nevts*ntrks);
   avgdx = avgdx/float(nevts*ntrks);
   avgdy = avgdy/float(nevts*ntrks);
   avgdz = avgdz/float(nevts*ntrks);

   float stdx = 0, stdy = 0, stdz = 0;
   float stddx = 0, stddy = 0, stddz = 0;
   for (size_t ie=0;ie<nevts;++ie) {
     for (size_t it=0;it<ntrks;++it) {
       float x_ = x(outtrk,ie,it);
       float y_ = y(outtrk,ie,it);
       float z_ = z(outtrk,ie,it);
       stdx += (x_-avgx)*(x_-avgx);
       stdy += (y_-avgy)*(y_-avgy);
       stdz += (z_-avgz)*(z_-avgz);
       float hx_ = x(hit,ie,it);
       float hy_ = y(hit,ie,it);
       float hz_ = z(hit,ie,it);
       stddx += ((x_-hx_)/x_-avgdx)*((x_-hx_)/x_-avgdx);
       stddy += ((y_-hy_)/y_-avgdy)*((y_-hy_)/y_-avgdy);
       stddz += ((z_-hz_)/z_-avgdz)*((z_-hz_)/z_-avgdz);
     }
   }

   stdx = sqrtf(stdx/float(nevts*ntrks));
   stdy = sqrtf(stdy/float(nevts*ntrks));
   stdz = sqrtf(stdz/float(nevts*ntrks));
   stddx = sqrtf(stddx/float(nevts*ntrks));
   stddy = sqrtf(stddy/float(nevts*ntrks));
   stddz = sqrtf(stddz/float(nevts*ntrks));

   printf("track x avg=%f std/avg=%f\n", avgx, fabs(stdx/avgx));
   printf("track y avg=%f std/avg=%f\n", avgy, fabs(stdy/avgy));
   printf("track z avg=%f std/avg=%f\n", avgz, fabs(stdz/avgz));
   printf("track dx/x avg=%f std=%f\n", avgdx, stddx);
   printf("track dy/y avg=%f std=%f\n", avgdy, stddy);
   printf("track dz/z avg=%f std=%f\n", avgdz, stddz);
   printf("track pt avg=%f\n", avgpt);
   printf("track phi avg=%f\n", avgphi);
   printf("track theta avg=%f\n", avgtheta);
	
   hipFree(trk);
   hipFree(hit);
   hipFree(outtrk);
   //free(trk);
   //free(hit);
   //free(outtrk);
   hipFree(trk_dev);
   hipFree(hit_dev);
   hipFree(outtrk_dev);
   
return 0;
}

